#include <vector>
#include <iostream>
#include <hipblas.h>
#include "include/device_vector.cuh"

#define real_t double


int main() {
    Context context;

    size_t k = 8;
    std::vector<real_t> bData{1.0, 2.0, 3.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,
                              6.0, 7.0, 8.0,};
    DeviceMatrix<real_t> B(context, k, bData, MatrixStorageMode::rowMajor);
    SvdFactoriser<real_t> svdEngine(context, B, true, false);
    std::cout << "status = " << svdEngine.factorise() << std::endl;

    /* ~~~ print results ~~~ */
    std::cout << "B = " << B;
    std::cout << "S = " << svdEngine.singularValues();
    std::cout << "V' = " << svdEngine.rightSingularVectors();
    auto U = svdEngine.leftSingularVectors();
    if (U) std::cout << "U = " << U.value();
    std::cout << "rank B = " << svdEngine.rank() << std::endl;

    return 0;
}
