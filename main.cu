#include <random>
#include <algorithm>
#include <iterator>
#include <vector>
#include <iostream>
#include <hipblas.h>
#include "include/tensor.cuh"
#include <memory>

#define real_t double



int main() {

    size_t m = 10;
    size_t n = 6;
    std::vector<double> v(m*n);
    v.reserve(m*n);
    std::iota(v.begin(), v.end(), 1);
    DTensor<double> a = DTensor<double>(v, m, n);

    auto ga = GivensAnnihilator<double>(a);
    ga.annihilate(0, 1, 2);

    std::cout << a;


    return 0;
}
