#include "hip/hip_runtime.h"
#include "include/tensor.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>


int main() {
    /* Write to binary file */
    auto r = DTensor<double>::createRandomTensor(3, 6, 4, -1, 1);
    std::string fName = "tensor.bt"; // binary tensor file extension: .bt
    r.saveToFile(fName);

    /* Parse binary file */
    auto recov = DTensor<double>::parseFromFile(fName);
    auto err = r - recov;
    std::cout << "max error : " << err.maxAbs();

    return 0;
}
