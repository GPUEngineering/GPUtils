#include "hip/hip_runtime.h"
#include "include/tensor.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

void xyz() {
    /* Write to binary file */
    auto r = DTensor<double>::createRandomTensor(3, 6, 4, -1, 1);
    auto r2 = DTensor<double>::createRandomTensor(300, 600, 4, -1, 1);
    std::string fName = "tensor.bt"; // binary tensor file extension: .bt

    /* Parse binary file */
    auto recov = DTensor<double>::parseFromFile(fName);
    auto err = r - recov;
    std::cout << "max error : " << err.maxAbs() << std::endl;
    std::cout << "Memory: " << std::setprecision(3)
            << (float) Session::getInstance().totalAllocatedBytes() / 1e6
            << " MB" << std::endl;
}


int main() {
    Session::getInstance(5);
    xyz();
    std::cout << "Memory (outside): " << std::setprecision(3)
            << (float) Session::getInstance().totalAllocatedBytes() / 1e6
            << " MB" << std::endl;
    return 0;
}
