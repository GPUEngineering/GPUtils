#include <random>
#include <algorithm>
#include <iterator>
#include <vector>
#include <iostream>
#include <hipblas.h>
#include "include/tensor.cuh"
#include <memory>

#define real_t double



int main() {
//    hipStream_t stream1;
//    hipStreamCreate(&stream1);
//    hipblasSetStream(Session::getInstance().cuBlasHandle(), stream1);

    hipStream_t s1;
    hipStreamCreate(&s1);

    auto a = DTensor<float>::createRandomTensor(2000, 200, 1, -2, 2);
    Svd svd(a);
    svd.factorise();

    std::cout << svd.singularValues();


    return 0;
}
