#include <vector>
#include <iostream>
#include <hipblas.h>
#include "include/device_vector.cuh"


int main() {
    Context context;

    size_t k = 8;
    std::vector<float> bData{1.0f, 2.0f, 3.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,};
    DeviceMatrix<float> B(&context, k, bData, MatrixStorageMode::rowMajor);
    SvdFactoriser<float> svdEngine(&context, B, true, false);
    svdEngine.factorise();

    /* ~~~ print results ~~~ */
    std::cout << "B = " << B;
    std::cout << "S = " << svdEngine.singularValues();
    std::cout << "V' = " << svdEngine.rightSingularVectors();
    auto U = svdEngine.leftSingularVectors();
    if (U.has_value()) {
        std::cout << "U = " << U.value();
    }
    return 0;
}
