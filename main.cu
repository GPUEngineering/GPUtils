#include <vector>
#include <iostream>
#include <hipblas.h>
#include "include/device.cuh"

#define real_t double


int main() {
    Context context;

    size_t kk = 3;
    std::vector<real_t> bbData{10.0, 2.0, 1.0,
                               2.0, 70.0, 1.5,
                               1.0, 1.5, 11.0};
    DeviceMatrix<real_t> B(context, kk, bbData, MatrixStorageMode::rowMajor);
    CholeskyFactoriser<real_t> choleskiser(context, B);
    std::cout << "status = " << choleskiser.factorise() << std::endl;
    std::cout << B;

    std::vector<real_t> bData = {-1.0, -2.0, 10.0};
    DeviceVector<real_t> b(context, bData);
    std::cout << "status = " <<  choleskiser.solve(b) << std::endl << std::endl;
    std::cout << b;
    return 0;
}
