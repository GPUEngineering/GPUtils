#include <vector>
#include <iostream>
#include <memory>
#include <hipblas.h>
#include <hipsolver.h>
#include "include/device_vector.cuh"
#include <iomanip>


int main() {
    Context context;

    size_t k = 4;
    std::vector<float> bData{1.0f, 2.0f, 3.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,
                             6.0f, 7.0f, 8.0f,};
    DeviceMatrix<float> B(&context, k, bData, MatrixStorageMode::rowMajor);

    DeviceMatrix<float> Bcopy(B);
    SvdFactoriser<float> svdEngine(&context, Bcopy, false);
    svdEngine.factorise();

    /* ~~~ print results ~~~ */
    std::cout << " B = " << B;
    std::cout << "S = " << *svdEngine.singularValues();
    std::cout << "V' = " << *svdEngine.rightSingularVectors();
    auto U = svdEngine.leftSingularVectors();
    if (U) std::cout << "U = " << *U;
//
//    std::vector<float> cData{1.5f, 2.0f, 3.0f,
//                             6.0f, 7.0f, 8.0f,
//                             6.0f, 7.0f, 8.0f,
//                             6.0f, 7.0f, 8.0f,};
//    DeviceMatrix<float> C(&context, k, cData, MatrixStorageMode::rowMajor);
//    svdEngine.updateMatrix(C);
//    svdEngine.factorise();
//
//    std::cout << "S = " << *svdEngine.singularValues();
//    if (svdEngine.leftSingularVectors()) {
//        std::cout << "U = " << *svdEngine.leftSingularVectors();
//    }


    return 0;
}
