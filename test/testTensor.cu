#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "../include/tensor.cuh"
#include <filesystem>

#define PRECISION_LOW 1e-4
#define PRECISION_HIGH 1e-10


/* ================================================================================================
 *  TENSOR<T> TESTS
 * ================================================================================================ */
class TensorTest : public testing::Test {
protected:
    TensorTest() {}

    virtual ~TensorTest() {}
};

#define TENSOR_DATA_234A {1, 2, 3, 4, 5, 6, 7, 8, 9, 8, 7, 10, 5, 4, 3, 2, 1, -1, 4, 3, 4, 3, 4, 8}
#define TENSOR_DATA_234B {7, -6, 9, 2, 1, 11, 34, -1, -4, -3, 12, 7, 9, 9, 2, 9, -9, -3, 2, 5, 4, -5, 4, 5}
#define TENSOR_DATA_234APB {8, -4, 12, 6, 6, 17, 41, 7, 5, 5, 19, 17, 14, 13, 5, 11, -8, -4, 6, 8, 8, -2, 8, 13}
#define TENSOR_DATA_234AMB {-6, 8, -6, 2, 4, -5, -27, 9, 13, 11, -5, 3, -4, -5, 1, -7, 10, 2, 2, -2, 0, 8, 0, 3};

/* ---------------------------------------
 * Zero Tensor (Constructor)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorConstructionZero() {
    DTensor<T> zero(2, 3, 4, true);
    EXPECT_EQ(2, zero.numRows());
    EXPECT_EQ(3, zero.numCols());
    EXPECT_EQ(4, zero.numMats());
    std::vector<T> expectedResult(2 * 3 * 4, 0);
    std::vector<T> zeroDown(2 * 3 * 4);
    zero.download(zeroDown);
    EXPECT_EQ(expectedResult, zeroDown);
}

TEST_F(TensorTest, tensorConstructionZero) {
    tensorConstructionZero<float>();
    tensorConstructionZero<double>();
    tensorConstructionZero<int>();
}

/* ---------------------------------------
 * Row- and column-major data
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorConstructionStorageMode() {
    size_t rows = 3;
    size_t cols = 2;
    size_t mats = 2;
    std::vector<T> aCm = {1, 3, 5,
                          2, 4, 6};
    std::vector<T> bCm = {7, 9, 11,
                          8, 10, 12};
    const std::vector<T> Cm = {1, 3, 5, 2, 4, 6, 7, 9, 11, 8, 10, 12};
    std::vector<T> aRm = {1, 2,
                          3, 4,
                          5, 6};
    std::vector<T> bRm = {7, 8,
                          9, 10,
                          11, 12};
    std::vector<T> Rm = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
    std::vector<T> hostData(rows * cols * mats);
    // test constructor
    DTensor<T> testCm(Cm, rows, cols, mats, columnMajor);
    DTensor<T> testRm(Rm, rows, cols, mats, rowMajor);
    testCm.download(hostData);
    EXPECT_EQ(Cm, hostData);
    testRm.download(hostData);
    EXPECT_EQ(Cm, hostData);
    // test .upload()
    DTensor<T> testSplitCm(rows, cols, mats);
    DTensor<T> ACm(testSplitCm, 2, 0, 0);
    DTensor<T> BCm(testSplitCm, 2, 1, 1);
    ACm.upload(aCm, columnMajor);
    BCm.upload(bCm, columnMajor);
    DTensor<T> testSplitRm(rows, cols, mats);
    DTensor<T> ARm(testSplitRm, 2, 0, 0);
    DTensor<T> BRm(testSplitRm, 2, 1, 1);
    ARm.upload(aRm, rowMajor);
    BRm.upload(bRm, rowMajor);
    testSplitCm.download(hostData);
    EXPECT_EQ(Cm, hostData);
    testSplitRm.download(hostData);
    EXPECT_EQ(Cm, hostData);
}

TEST_F(TensorTest, tensorConstructionStorageMode) {
    tensorConstructionStorageMode<float>();
    tensorConstructionStorageMode<double>();
    tensorConstructionStorageMode<int>();
}

/* ---------------------------------------
 * Create random tensors
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void randomTensorCreation() {
    // 20 x 40 x 60-tensor; elements drawn from U[-1, 1]
    auto r = DTensor<T>::createRandomTensor(20, 40, 60, -1, 1);
    EXPECT_EQ(20, r.numRows());
    EXPECT_EQ(40, r.numCols());
    EXPECT_EQ(60, r.numMats());
    auto rEle = r(19, 39, 59);
    EXPECT_TRUE(rEle >= -1 && rEle <= 1);
}

TEST_F(TensorTest, randomTensorCreation) {
    randomTensorCreation<float>();
    randomTensorCreation<double>();
    randomTensorCreation<int>();
}

/* ---------------------------------------
 * Save to file and parse
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void parseTensorFromFile() {
    size_t nR = 20, nC = 40, nM = 60;
    auto r = DTensor<T>::createRandomTensor(nR, nC, nM, -1, 1);
    std::string fName = "myTest.dtensor";
    r.saveToFile(fName);
    auto a = DTensor<T>::parseFromTextFile(fName);
    EXPECT_EQ(nR, a.numRows());
    EXPECT_EQ(nC, a.numCols());
    EXPECT_EQ(nM, a.numMats());
    auto diff = a - r;
    T err = diff.maxAbs();
    EXPECT_LT(err, 1e-15);
}

TEST_F(TensorTest, parseTensorFromFile) {
    parseTensorFromFile<float>();
    parseTensorFromFile<double>();
}

/* ---------------------------------------
 * Move constructor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorMoveConstructor() {
    DTensor<T> zero(2, 3, 4, true);
    DTensor<T> x(std::move(zero));
    DTensor<T> y(DTensor<T> {100, 10, 1000});
}

TEST_F(TensorTest, tensorMoveConstructor) {
    tensorMoveConstructor<float>();
    tensorMoveConstructor<double>();
    tensorMoveConstructor<int>();
    tensorMoveConstructor<int *>();
    tensorMoveConstructor<double *>();
}

/* ---------------------------------------
 * New tensor from data (std::vector)
 * Constructor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorConstructionFromVector() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    EXPECT_EQ(2, tenz.numRows());
    EXPECT_EQ(3, tenz.numCols());
    EXPECT_EQ(4, tenz.numMats());
    EXPECT_EQ(2 * 3 * 4, tenz.numEl());
}

TEST_F(TensorTest, tensorConstructionFromVector) {
    tensorConstructionFromVector<float>();
    tensorConstructionFromVector<double>();
    tensorConstructionFromVector<int>();
}

/* ---------------------------------------
 * Tensor: Copy constructor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorCopyConstructor() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    DTensor<T> tenzCp(tenz);
    EXPECT_EQ(2, tenzCp.numRows());
    EXPECT_EQ(3, tenzCp.numCols());
    EXPECT_EQ(4, tenzCp.numMats());
    EXPECT_EQ(2 * 3 * 4, tenzCp.numEl());
    std::vector<T> tenzDown(2 * 3 * 4);
    tenzCp.download(tenzDown);
    EXPECT_EQ(data, tenzDown);
    EXPECT_NE(tenz.raw(), tenzCp.raw());
}

TEST_F(TensorTest, tensorCopyConstructor) {
    tensorCopyConstructor<float>();
    tensorCopyConstructor<double>();
    tensorCopyConstructor<int>();
}

/* ---------------------------------------
 * Tensor: Slicing constructor
 * axis = 2 (matrices)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSlicingConstructorAxis2() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tens(data, 2, 3, 4);
    DTensor<T> tensSlice(tens, 2, 0, 1); // matrices #0 and #1
    EXPECT_EQ(2, tensSlice.numRows());
    EXPECT_EQ(3, tensSlice.numCols());
    EXPECT_EQ(2, tensSlice.numMats());
    EXPECT_EQ(tens.raw(), tensSlice.raw()); // it is indeed a slice
    EXPECT_TRUE(tensSlice.ptrMatrices() != nullptr);
}

TEST_F(TensorTest, tensorSlicingConstructorAxis2) {
    tensorSlicingConstructorAxis2<float>();
    tensorSlicingConstructorAxis2<double>();
    tensorSlicingConstructorAxis2<int>();
}

/* ---------------------------------------
 * Tensor: Slicing constructor
 * axis = 1 (columns)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSlicingConstructorAxis1() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    DTensor<T> tenzSlice(tenz, 1, 1, 2); // columns from 1 to 2
    EXPECT_EQ(2, tenzSlice.numRows());
    EXPECT_EQ(2, tenzSlice.numCols());
    EXPECT_EQ(1, tenzSlice.numMats());
    EXPECT_TRUE(tenzSlice.ptrMatrices() == nullptr);
    std::vector<T> expected = {3, 4, 5, 6};
    std::vector<T> tenzSliceDown(4);
    tenzSlice.download(tenzSliceDown);
    EXPECT_EQ(expected, tenzSliceDown);
}

TEST_F(TensorTest, tensorSlicingConstructorAxis1) {
    tensorSlicingConstructorAxis1<float>();
    tensorSlicingConstructorAxis1<double>();
    tensorSlicingConstructorAxis1<int>();
}

/* ---------------------------------------
 * Tensor: Slicing constructor
 * axis = 0 (rows)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSlicingConstructorAxis0() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    DTensor<T> tenzSlice(tenz, 0, 2, 3); // elements 2..3
    EXPECT_EQ(2, tenzSlice.numRows());
    EXPECT_EQ(1, tenzSlice.numCols());
    EXPECT_EQ(1, tenzSlice.numMats());
    EXPECT_TRUE(tenzSlice.ptrMatrices() == nullptr);
    std::vector<T> expected = {3, 4};
    std::vector<T> tenzSliceDown(2);
    tenzSlice.download(tenzSliceDown);
    EXPECT_EQ(expected, tenzSliceDown);
}

TEST_F(TensorTest, tensorSlicingConstructorAxis0) {
    tensorSlicingConstructorAxis0<float>();
    tensorSlicingConstructorAxis0<double>();
    tensorSlicingConstructorAxis0<int>();
}

/* ---------------------------------------
 * Tensor: Upload data
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorUpload() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(2, 3, 4);
    tenz.upload(data);
    EXPECT_EQ(2, tenz.numRows());
    EXPECT_EQ(3, tenz.numCols());
    EXPECT_EQ(4, tenz.numMats());
    EXPECT_EQ(2 * 3 * 4, tenz.numEl());
    EXPECT_EQ(4, tenz.numMats());
    EXPECT_EQ(8, tenz(1, 2, 3));
}

TEST_F(TensorTest, tensorUpload) {
    tensorUpload<float>();
    tensorUpload<double>();
    tensorUpload<int>();
}

/* ---------------------------------------
 * Tensor: deviceCopyTo
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorDeviceCopyTo() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    DTensor<T> other(2, 3, 5, true);
    DTensor<T> z(other, 2, 1, 4);
    tenz.deviceCopyTo(z);
    std::vector<T> expected = {0, 0, 0, 0, 0, 0,
                               1, 2, 3, 4, 5, 6, 7, 8, 9, 8, 7, 10, 5, 4, 3, 2, 1, -1, 4, 3, 4, 3, 4, 8};
    std::vector<T> actual(2 * 3 * 5);
    other.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorDeviceCopyTo) {
    tensorDeviceCopyTo<float>();
    tensorDeviceCopyTo<double>();
    tensorDeviceCopyTo<int>();
}


/* ---------------------------------------
 * Tensor: Reshape
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorReshape() {
    size_t m = 5, n = 10, k = 3;
    DTensor<T> a = DTensor<T>::createRandomTensor(m, n, k, -1, 1); // dim = (m, n, k)
    T lastElement = a(m - 1, n - 1, k - 1); // last element
    T firstElement = a(0, 0, 0);
    ASSERT_EQ(m, a.numRows());
    ASSERT_EQ(n, a.numCols());
    ASSERT_EQ(k, a.numMats());
    a.reshape(m, k, n); // dim = (m, k, n)
    ASSERT_EQ(m, a.numRows());
    ASSERT_EQ(k, a.numCols());
    ASSERT_EQ(n, a.numMats());
    a.reshape(k, n, m); // dim = (k, n, m)
    ASSERT_EQ(k, a.numRows());
    ASSERT_EQ(n, a.numCols());
    ASSERT_EQ(m, a.numMats());
    a.reshape(k * n, m, 1); // dim = (k*n, m, 1)
    ASSERT_EQ(k * n, a.numRows());
    ASSERT_EQ(m, a.numCols());
    ASSERT_EQ(1, a.numMats());
    a.reshape(m, k * n, 1); // dim = (m, k*n, 1)
    ASSERT_EQ(m, a.numRows());
    ASSERT_EQ(k * n, a.numCols());
    ASSERT_EQ(1, a.numMats());
    a.reshape(m * k * n, 1, 1); // dim = (m*k*n, 1, 1)
    ASSERT_EQ(m * k * n, a.numRows());
    ASSERT_EQ(1, a.numCols());
    ASSERT_EQ(1, a.numMats());
    ASSERT_EQ(lastElement, a(m * n * k - 1, 0, 0));
    ASSERT_EQ(firstElement, a(0, 0, 0));
}

TEST_F(TensorTest, tensorReshape) {
    tensorReshape<float>();
    tensorReshape<double>();
    tensorReshape<int>();
}

/* ---------------------------------------
 * Tensor: Slice, reshape and add/multiply
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSliceAndReshape(T epsilon) {
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> a(dataA, 2, 3, 4);
    DTensor<T> b(dataB, 2, 3, 4);

    /* ---- Slicing axis = 2 ---- */
    DTensor<T> aSlice(a, 2, 1, 3);
    DTensor<T> bSlice(b, 2, 1, 3);
    aSlice.reshape(2, 9, 1);
    bSlice.reshape(2, 9, 1);
    aSlice += bSlice;

    std::vector<T> dataAExpected = {1, 2, 3, 4, 5, 6, 41, 7, 5, 5,
                                    19, 17, 14, 13, 5, 11, -8, -4,
                                    6, 8, 8, -2, 8, 13};
    DTensor<T> aExpected(dataAExpected, 2, 3, 4);

    DTensor<T> err = aExpected - a;
    ASSERT_LT(err.normF(), epsilon);
}

TEST_F(TensorTest, tensorSliceAndReshape) {
    tensorSliceAndReshape<float>(PRECISION_LOW);
    tensorSliceAndReshape<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Tensor: Frobenius dot product
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorDotF(T epsilon) {
    // as vectors
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> vecA(dataA, dataA.size());
    DTensor<T> vecB(dataB, dataB.size());
    T dotVector = vecA.dotF(vecB);
    EXPECT_EQ(604, dotVector);  // from MATLAB
    // as matrices
    DTensor<T> tenA(dataA, 2, 3, 4);
    DTensor<T> tenB(dataB, 2, 3, 4);
    T dotTensor = tenA.dotF(tenB);
    EXPECT_EQ(604, dotTensor);  // from MATLAB
}

TEST_F(TensorTest, tensorDotF) {
    tensorDotF<float>(PRECISION_LOW);
    tensorDotF<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Tensor: Frobenius norm
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorNormF(T epsilon) {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    EXPECT_NEAR(26.153393661244042, tenz.normF(), epsilon); // from MATLAB
}

TEST_F(TensorTest, tensorNormF) {
    tensorNormF<float>(PRECISION_LOW);
    tensorNormF<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Tensor: sum of absolute value of
 * all elements
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSumAbs() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    EXPECT_NEAR(112, tenz.sumAbs(), PRECISION_HIGH); // from MATLAB
}

TEST_F(TensorTest, tensorSumAbs) {
    tensorSumAbs<float>();
    tensorSumAbs<double>();
}

/* ---------------------------------------
 * Tensor: max of absolute of all elements
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorMax() {
    std::vector<T> data = TENSOR_DATA_234AMB;
    DTensor<T> tenz(data, 2, 3, 4);
    T m = tenz.maxAbs();
    EXPECT_EQ(27, m);
}

TEST_F(TensorTest, tensorMax) {
    tensorMax<float>();
    tensorMax<double>();
}

/* ---------------------------------------
 * Tensor: min of absolute of all elements
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorMin() {
    std::vector<T> data = TENSOR_DATA_234AMB;
    DTensor<T> tenz(data, 2, 3, 4);
    T m = tenz.minAbs();
    EXPECT_EQ(0, m);
}

TEST_F(TensorTest, tensorMin) {
    tensorMin<float>();
    tensorMin<double>();
}

/* ---------------------------------------
 * Tensor: right Givens rotation
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorRightGivens(T epsilon) {
    // Construct matrix A
    size_t m = 10;
    size_t n = 6;
    std::vector<T> v(m * n);
    v.reserve(m * n);
    std::iota(v.begin(), v.end(), 1);
    auto a = DTensor<T>(v, m, n, 1);

    // Apply right Givens rotation G
    size_t i_givens = 1, j_givens = 4;
    T c = 0.1;
    T minus_s = sqrt(1 - c * c);
    a.applyRightGivensRotation(i_givens, j_givens, &c, &minus_s);

    // Check the result
    for (size_t i = 0; i < m; i++) {
        EXPECT_NEAR(1 + i, a(i, 0), epsilon);
        EXPECT_NEAR(21 + i, a(i, 2), epsilon);
        EXPECT_NEAR(31 + i, a(i, 3), epsilon);
        EXPECT_NEAR((11 + i) * c - (41 + i) * (-minus_s), a(i, i_givens), epsilon);
        EXPECT_NEAR((11 + i) * (-minus_s) + (41 + i) * c, a(i, j_givens), epsilon);
    }
}

TEST_F(TensorTest, tensorRightGivens) {
    tensorRightGivens<float>(PRECISION_LOW);
    tensorRightGivens<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Tensor: left Givens rotation
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorLeftGivens(T epsilon) {
    // Construct matrix A
    size_t m = 10;
    size_t n = 6;
    std::vector<double> v(m * n);
    v.reserve(m * n);
    std::iota(v.begin(), v.end(), 1);
    auto a = DTensor<double>(v, m, n, 1);

    // Apply right Givens rotation G
    size_t i_givens = 1, j_givens = 9;
    double c = 0.1;
    double minus_s = -sqrt(1 - c * c);
    a.applyLeftGivensRotation(i_givens, j_givens, &c, &minus_s);


    // Check the result
    for (size_t j = 0; j < n; j++) {
        EXPECT_NEAR(1 + 10 * j, a(0, j), epsilon);
        for (size_t i = 2; i < m - 1; i++) {
            EXPECT_NEAR(1 + i + 10 * j, a(i, j), epsilon);
        }
        EXPECT_NEAR((2 + 10 * j) * c + (10 + 10 * j) * minus_s, a(i_givens, j), epsilon);
        EXPECT_NEAR((2 + 10 * j) * (-minus_s) + (10 + 10 * j) * c, a(j_givens, j), epsilon);
    }
}

TEST_F(TensorTest, tensorLeftGivens) {
    tensorLeftGivens<float>(1e-10);
    tensorLeftGivens<double>(1e-14);
}

/* ---------------------------------------
 * Tensor operator() to access element
 * e.g., t(2, 3, 4)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorBracketOperator() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    EXPECT_EQ(1, tenz(0, 0, 0));
    EXPECT_EQ(3, tenz(0, 1, 2));
    EXPECT_EQ(8, tenz(1, 2, 3));
}

TEST_F(TensorTest, tensorBracketOperator) {
    tensorBracketOperator<float>();
    tensorBracketOperator<double>();
    tensorBracketOperator<int>();
}

/* ---------------------------------------
 * Tensor assignment operator
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorAssignmentOperator() {
    std::vector<T> data = TENSOR_DATA_234A;
    DTensor<T> tenz(data, 2, 3, 4);
    DTensor<T> other;
    other = tenz;
    EXPECT_EQ(tenz.raw(), other.raw());
    EXPECT_EQ(2, other.numRows());
    EXPECT_EQ(3, other.numCols());
    EXPECT_EQ(4, other.numMats());
}

TEST_F(TensorTest, tensorAssignmentOperator) {
    tensorAssignmentOperator<float>();
    tensorAssignmentOperator<double>();
    tensorAssignmentOperator<int>();
}

/* ---------------------------------------
 * Tensor times-equals scalar
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorTimesEqualsScalar() {
    std::vector<T> data = TENSOR_DATA_234A;
    std::vector<T> dataTimes3 = {3, 6, 9, 12, 15, 18, 21, 24, 27, 24, 21, 30, 15, 12, 9, 6, 3, -3, 12, 9, 12, 9, 12,
                                 24};
    DTensor<T> tenz(data, 2, 3, 4);
    tenz *= 3.0;
    std::vector<T> actual;
    tenz.download(actual);
    EXPECT_EQ(dataTimes3, actual);
}

TEST_F(TensorTest, tensorTimesEqualsScalar) {
    tensorTimesEqualsScalar<float>();
    tensorTimesEqualsScalar<double>();
}

/* ---------------------------------------
 * Scalar times tensor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorTimesScalar() {
    std::vector<T> data = TENSOR_DATA_234A;
    std::vector<T> dataTimes3 = {3, 6, 9, 12, 15, 18, 21, 24, 27, 24, 21, 30, 15, 12, 9, 6, 3, -3, 12, 9, 12, 9, 12,
                                 24};
    DTensor<T> tenz(data, 2, 3, 4);
    auto tripleTensor = 3.0 * tenz;
    std::vector<T> actual;
    tripleTensor.download(actual);
    EXPECT_EQ(dataTimes3, actual);
}

TEST_F(TensorTest, tensorTimesScalar) {
    tensorTimesScalar<float>();
    tensorTimesScalar<double>();
}

/* ---------------------------------------
 * Tensor plus-equals tensor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorPlusEqualsTensor() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> A(dataA, 2, 3, 4);
    DTensor<T> B(dataB, 2, 3, 4);
    A += B;
    std::vector<T> expected = TENSOR_DATA_234APB;
    std::vector<T> actual;
    A.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorPlusEqualsTensor) {
    tensorPlusEqualsTensor<float>();
    tensorPlusEqualsTensor<double>();
}

/* ---------------------------------------
 * Tensor minus-equals tensor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorMinusEqualsTensor() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> A(dataA, 2, 3, 4);
    DTensor<T> B(dataB, 2, 3, 4);
    A -= B;
    std::vector<T> expected = TENSOR_DATA_234AMB;
    std::vector<T> actual;
    A.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorMinusEqualsTensor) {
    tensorMinusEqualsTensor<float>();
    tensorMinusEqualsTensor<double>();
}

/* ---------------------------------------
 * Tensor + Tensor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorPlusTensor() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> A(dataA, 2, 3, 4);
    DTensor<T> B(dataB, 2, 3, 4);
    DTensor<T> C = A + B;
    std::vector<T> expected = TENSOR_DATA_234APB;
    std::vector<T> actual;
    C.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorPlusTensor) {
    tensorPlusTensor<float>();
    tensorPlusTensor<double>();
}

/* ---------------------------------------
 * Tensor - Tensor
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorMinusTensor() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    std::vector<T> dataB = TENSOR_DATA_234B;
    DTensor<T> A(dataA, 2, 3, 4);
    DTensor<T> B(dataB, 2, 3, 4);
    DTensor<T> C = A - B;
    std::vector<T> expected = TENSOR_DATA_234AMB;
    std::vector<T> actual;
    C.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorMinusTensor) {
    tensorMinusTensor<float>();
    tensorMinusTensor<double>();
}

/* ---------------------------------------
 * Tensor: C = AB
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorAddAB() {
    std::vector<T> aData = {1, 2, 3, 4, 5, 6,
                            7, 8, 9, 10, 11, 12,
                            13, 14, 15, 16, 17, 18};
    std::vector<T> bData = {6, 5, 4, 3, 2, 1,
                            7, 6, 5, 4, 3, 2,
                            1, 2, 1, 5, -6, 8};
    DTensor<T> A(aData, 2, 3, 3);
    DTensor<T> B(bData, 3, 2, 3);
    DTensor<T> C(2, 2, 3, true);
    C.addAB(A, B);
    std::vector<T> expected = {41, 56, 14, 20, 158, 176, 77, 86, 60, 64, 111, 118};
    std::vector<T> actual;
    C.download(actual);
    EXPECT_EQ(expected, actual);
}

TEST_F(TensorTest, tensorAddAB) {
    tensorAddAB<double>();
    tensorAddAB<float>();
}

/* ---------------------------------------
 * Tensor: slice ptrMatrices
 * axis = 2 (matrices)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSliceAxis2PtrMatrices() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    DTensor<T> d_A(dataA, 2, 3, 4);
    DTensor<T> d_ASlice(d_A, 2, 2, 3);
    EXPECT_TRUE(d_ASlice.ptrMatrices() == d_A.ptrMatrices() + 2);
}

TEST_F(TensorTest, tensorSliceAxis2PtrMatrices) {
    tensorSliceAxis2PtrMatrices<float>();
    tensorSliceAxis2PtrMatrices<double>();
    tensorSliceAxis2PtrMatrices<int>();
}

/* ---------------------------------------
 * Tensor: slice ptrMatrices
 * axis = 0 and 1
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorSliceAxis01PtrMatrices() {
    std::vector<T> dataA = TENSOR_DATA_234A;
    DTensor<T> d_A(dataA, 2, 3, 4);
    DTensor<T> d_ASlice0(d_A, 0, 0, 1);
    EXPECT_TRUE(!d_ASlice0.ptrMatrices());
    DTensor<T> d_ASlice1(d_A, 1, 0, 2);
    EXPECT_TRUE(!d_ASlice0.ptrMatrices());
}

TEST_F(TensorTest, tensorSliceAxis01PtrMatrices) {
    tensorSliceAxis01PtrMatrices<float>();
    tensorSliceAxis01PtrMatrices<double>();
    tensorSliceAxis01PtrMatrices<int>();
}

/* ---------------------------------------
 * Tensor: getRows
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorGetRows() {
    std::vector<T> aData = {10.5, 25.0, 60.0,
                            -21.0, 720.0, -1.0,
                            11.0, -1.0, 30.0,
                            5., 6., 7.,
                            8., 9., 10.,
                            11., 12., 13};
    DTensor<T> A(aData, 3, 3, 2);
    DTensor<T> Ar0 = A.getRows(1, 1, 0);
    std::vector<T> expected0 = {25., 720., -1.};
    std::vector<T> actual0(3);
    Ar0.download(actual0);
    EXPECT_EQ(expected0, actual0);

    DTensor<T> Ar1 = A.getRows(1, 2, 1);
    std::vector<T> expected1 = {6., 7., 9., 10., 12., 13.};
    std::vector<T> actual1(6);
    Ar1.download(actual1);
    EXPECT_EQ(expected1, actual1);
}

TEST_F(TensorTest, tensorGetRows) {
    tensorGetRows<float>();
    tensorGetRows<double>();
}


/* ---------------------------------------
 * Tensor: transpose
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorTranspose() {
    std::vector<T> aData = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
    DTensor<T> A(aData, 3, 2, 2);
    DTensor<T> Atranspose = A.tr();
    EXPECT_EQ(2, Atranspose.numRows());
    EXPECT_EQ(3, Atranspose.numCols());
    EXPECT_EQ(2, Atranspose.numMats());
    std::vector<T> expected = {1, 4, 2, 5, 3, 6, 7, 10, 8, 11, 9, 12};
    std::vector<T> actual;
    Atranspose.download(actual);
    EXPECT_EQ(expected, actual);

}

TEST_F(TensorTest, tensorTranspose) {
    tensorTranspose<float>();
    tensorTranspose<double>();
}

/* ================================================================================================
 *  LEAST SQUARES TESTS
 * ================================================================================================ */
class LeastSquaresTest : public testing::Test {
protected:
    LeastSquaresTest() {}

    virtual ~LeastSquaresTest() {}
};

/* ---------------------------------------
 * Tensor: Least squares
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T
void tensorLeastSquares1(T epsilon) {
    // TODO test with tall matrices too
    std::vector<T> aData = {1, 2,
                            3, 4,
                            7, 8,
                            9, 10,
                            6, 8,
                            -9, 20};
    std::vector<T> bData = {1, 1, -1, 2, 30, -80};
    DTensor<T> A0(aData, 2, 2, 3);
    DTensor<T> A(A0);
    DTensor<T> B(bData, 2, 1, 3);
    DTensor<T> sol(B);
    A0.leastSquaresBatched(sol);
    DTensor<T> C(2, 1, 3);
    C.addAB(A, sol);
    C -= B;
    T nrmErr = C.normF();
    EXPECT_LT(nrmErr, epsilon);
}

TEST_F(LeastSquaresTest, tensorLS1) {
    tensorLeastSquares1<float>(PRECISION_LOW);
    tensorLeastSquares1<double>(PRECISION_HIGH);
}


/* ================================================================================================
 *  SVD TESTS
 * ================================================================================================ */
class SvdTest : public testing::Test {
protected:
    SvdTest() {}

    virtual ~SvdTest() {}
};

/* ---------------------------------------
 * Computation of singular values
 * and matrix rank
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void singularValuesComputation(float epsilon) {
    std::vector<T> bData = {1, 6, 6, 6, 6, 6, 6, 6,
                            2, 7, 7, 7, 7, 7, 7, 7,
                            3, 8, 8, 8, 8, 8, 8, 8,};
    DTensor<T> B(bData, 8, 3);
    Svd<T> svd(B, true, false);
    svd.factorise();
    for (size_t i = 0; i < B.numMats(); i++) {
        EXPECT_EQ(0, svd.info()(i));
    }
    auto S = svd.singularValues();
    EXPECT_NEAR(32.496241123753592, S(0), epsilon); // value from MATLAB
    EXPECT_NEAR(0.997152358903242, S(1), epsilon); // value from MATLAB

    auto U = svd.leftSingularVectors();
    EXPECT_TRUE(U.has_value());
}

TEST_F(SvdTest, singularValuesComputation) {
    singularValuesComputation<float>(PRECISION_LOW);
    singularValuesComputation<double>(PRECISION_HIGH);
}


/* ---------------------------------------
 * Singular values - memory mgmt
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void singularValuesMemory(float epsilon) {
    std::vector<T> bData = {1, 6, 6, 6, 6, 6, 6, 6,
                            2, 7, 7, 7, 7, 7, 7, 7,
                            3, 8, 8, 8, 8, 8, 8, 8,};
    DTensor<T> B(bData, 8, 3);
    Svd<T> svd(B, true, false);
    svd.factorise();
    EXPECT_EQ(0, svd.info()(0));
    DTensor<T> const &v1 = svd.rightSingularVectors();
    DTensor<T> const &v2 = svd.rightSingularVectors();
    EXPECT_EQ(&v1, &v2);
    EXPECT_EQ(v1.raw(), v2.raw());
    DTensor<T> const &s1 = svd.singularValues();
    DTensor<T> const &s2 = svd.singularValues();
    EXPECT_EQ(&s1, &s2);
    EXPECT_EQ(s1.raw(), s2.raw());
    auto u1 = svd.leftSingularVectors().value();
    auto u2 = svd.leftSingularVectors().value();
    EXPECT_EQ(u1, u2);
    EXPECT_EQ(u1->raw(), u2->raw());
}

TEST_F(SvdTest, singularValuesMemory) {
    singularValuesMemory<float>(PRECISION_LOW);
    singularValuesMemory<double>(PRECISION_HIGH);
}


/* ---------------------------------------
 * SVD with multiple matrices
 * --------------------------------------- */
TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void singularValuesMultipleMatrices(float epsilon) {
    std::vector<T> aData = {1, 2, 3, 4, 5, 6, 1, 1, 1, 2, 2, 2, 0, 0, 0, 0, 0, 1};
    DTensor<T> A(aData, 3, 2, 3);
    Svd<T> svd(A, true); // do compute U (A will be destroyed)
    svd.factorise();
    DTensor<T> const &S = svd.singularValues();
    DTensor<T> const &V = svd.rightSingularVectors();
    auto Uopt = svd.leftSingularVectors();
    auto U = Uopt.value();
    std::vector<T> expected_v = {-0.386317703118612, -0.922365780077058, -0.922365780077058, 0.386317703118612,
                                 -0.447213595499958, -0.894427190999916, 0.894427190999916, -0.447213595499958,
                                 0, -1, 1, 0};
    std::vector<T> actual_v(12);
    V.download(actual_v);
    for (size_t i = 0; i < 4; i++) EXPECT_NEAR(expected_v[i], actual_v[i], epsilon);
    std::vector<T> expected_s = {9.508032000695726, 0.772869635673484, 3.872983346207417, 0, 1, 0};
    std::vector<T> actual_s(6);
    S.download(actual_s);
    for (size_t i = 0; i < 6; i++) EXPECT_NEAR(expected_s[i], actual_s[i], epsilon);
    std::vector<T> expected_u = {
            -0.428667133548626, -0.566306918848035, -0.703946704147444,
            0.805963908589298, 0.112382414096594, -0.581199080396110,
            0.408248290463863, -0.816496580927726, 0.408248290463863,
            -0.577350269189626, -0.577350269189626, -0.577350269189626,
            0.816496580927726, -0.408248290463863, -0.408248290463863,
            0.000000000000000, -0.707106781186548, 0.707106781186547,
            0, 0, -1,
            1, 0, 0,
            0, -1, 0,
    };
    std::vector<T> actual_u(27);
    U->download(actual_u);
    for (size_t i = 0; i < 27; i++) EXPECT_NEAR(expected_u[i], actual_u[i], epsilon);

}

TEST_F(SvdTest, singularValuesMultipleMatrices) {
    singularValuesMultipleMatrices<float>(10 * PRECISION_LOW); // SVD with float performs quite poorly
    singularValuesMultipleMatrices<double>(PRECISION_HIGH);
}


/* ---------------------------------------
 * SVD for rank computation of multiple
 * matrices
 * --------------------------------------- */
TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void singularValuesRankMultipleMatrices(float epsilon) {
    std::vector<T> aData = {1, 4, 7, 10, 2, 5, 8, 11, 3, 6, 9, 0,
                            1, 4, 7, 10, 2, 5, 8, 11, 3, 6, 9, 12,
                            1, 2, 3, 4, 2, 4, 6, 8, 3, 6, 9, 12};
    DTensor<T> A(aData, 4, 3, 3);

    Svd<T> svd(A);
    svd.factorise();
    auto rank = svd.rank(epsilon);
    EXPECT_EQ(3, rank(0, 0, 0));
    EXPECT_EQ(2, rank(0, 0, 1));
    EXPECT_EQ(1, rank(0, 0, 2));
}

TEST_F(SvdTest, singularValuesRankMultipleMatrices) {
    singularValuesRankMultipleMatrices<float>(PRECISION_LOW); // SVD with float performs quite poorly
    singularValuesRankMultipleMatrices<double>(PRECISION_HIGH);
}

/* ================================================================================================
 *  CHOLESKY TESTS
 * ================================================================================================ */
class CholeskyTest : public testing::Test {
protected:
    CholeskyTest() {}

    virtual ~CholeskyTest() {}
};


/* ---------------------------------------
 * Cholesky factorisation
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void choleskyFactorisation(T epsilon) {
    std::vector<T> aData = {10.0, 2.0, 3.0,
                            2.0, 20.0, -1.0,
                            3.0, -1.0, 30.0};
    DTensor<T> A(aData, 3, 3, 1);
    CholeskyFactoriser<T> chol(A);
    chol.factorise();
    EXPECT_NEAR(3.162277660168380, A(0, 0), epsilon);
    EXPECT_NEAR(-0.361403161162101, A(2, 1), epsilon);
    EXPECT_NEAR(5.382321781081287, A(2, 2), epsilon);
    EXPECT_EQ(0, chol.info()(0));
}

TEST_F(CholeskyTest, choleskyFactorisation) {
    choleskyFactorisation<float>(PRECISION_LOW);
    choleskyFactorisation<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Cholesky factorisation: solve system
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void choleskyFactorisationSolution(T epsilon) {
    std::vector<T> aData = {10.0, 2.0, 3.0,
                            2.0, 20.0, -1.0,
                            3.0, -1.0, 30.0};
    DTensor<T> A(aData, 3, 3, 1);
    DTensor<T> L(A); // L = A
    CholeskyFactoriser<T> chol(L);
    chol.factorise();

    std::vector<T> bData = {-1., -3., 5.};
    DTensor<T> rhs(bData, 3, 1, 1);
    DTensor<T> sol(rhs);
    chol.solve(sol);

    std::vector<T> expected = {-0.126805213103205, -0.128566396618528, 0.175061641423036};
    std::vector<T> actual(3);
    sol.download(actual);
    for (size_t i = 0; i < 3; i++) EXPECT_NEAR(expected[i], actual[i], epsilon);

    DTensor<T> error = A * sol;
    error -= rhs;
    EXPECT_TRUE(error.normF() < epsilon);

}

TEST_F(CholeskyTest, choleskyFactorisationSolution) {
    choleskyFactorisationSolution<float>(PRECISION_LOW);
    choleskyFactorisationSolution<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Batched Cholesky factorisation
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void choleskyBatchFactorisation(T epsilon) {
    std::vector<T> aData = {10.0, 2.0, 3.0,
                            2.0, 20.0, -1.0,
                            3.0, -1.0, 30.0};
    DTensor<T> A(3, 3, 2);
    DTensor<T> A0(A, 2, 0, 0);
    DTensor<T> A1(A, 2, 1, 1);
    A0.upload(aData);
    A1.upload(aData);
    CholeskyBatchFactoriser<T> chol(A);
    chol.factorise();
    // 0
    EXPECT_NEAR(3.162277660168380, A(0, 0, 0), epsilon);
    EXPECT_NEAR(-0.361403161162101, A(2, 1, 0), epsilon);
    EXPECT_NEAR(5.382321781081287, A(2, 2, 0), epsilon);
    // 1
    EXPECT_NEAR(3.162277660168380, A(0, 0, 1), epsilon);
    EXPECT_NEAR(-0.361403161162101, A(2, 1, 1), epsilon);
    EXPECT_NEAR(5.382321781081287, A(2, 2, 1), epsilon);

    EXPECT_EQ(0, chol.info()(0));
}

TEST_F(CholeskyTest, choleskyBatchFactorisation) {
    choleskyBatchFactorisation<float>(PRECISION_LOW);
    choleskyBatchFactorisation<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Batched Cholesky solve
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void choleskyBatchFactorSolve(T epsilon) {
    std::vector<T> aData = {10.0, 2.0, 3.0,
                            2.0, 20.0, -1.0,
                            3.0, -1.0, 30.0};
    DTensor<T> A(3, 3, 2);
    DTensor<T> A0(A, 2, 0, 0);
    DTensor<T> A1(A, 2, 1, 1);
    A0.upload(aData);
    A1.upload(aData);
    DTensor<T> L(A); // L = A
    CholeskyBatchFactoriser<T> chol(L);
    chol.factorise();
    std::vector<T> bData = {-1., -3., 5.};
    DTensor<T> rhs(3, 1, 2);
    DTensor<T> rhs0(rhs, 2, 0, 0);
    DTensor<T> rhs1(rhs, 2, 1, 1);
    rhs0.upload(bData);
    rhs1.upload(bData);
    DTensor<T> sol(rhs);
    chol.solve(sol);
    std::vector<T> expected = {-0.126805213103205, -0.128566396618528, 0.175061641423036};
    std::vector<T> actual(6);
    sol.download(actual);
    for (size_t i = 0; i < 3; i++) EXPECT_NEAR(expected[i], actual[i], epsilon);  // 0
    for (size_t i = 0; i < 3; i++) EXPECT_NEAR(expected[i], actual[i + 3], epsilon);  // 1
    DTensor<T> error = A * sol;
    error -= rhs;
    EXPECT_TRUE(error.normF() < epsilon);
    EXPECT_EQ(0, chol.info()(0));
}

TEST_F(CholeskyTest, choleskyBatchFactorSolve) {
    choleskyBatchFactorSolve<float>(PRECISION_LOW);
    choleskyBatchFactorSolve<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Batched Cholesky solve (factor provided)
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void choleskyBatchSolve(T epsilon) {
    std::vector<T> aData = {10.0, 2.0, 3.0,
                            2.0, 20.0, -1.0,
                            3.0, -1.0, 30.0};
    DTensor<T> A(3, 3, 2);
    DTensor<T> A0(A, 2, 0, 0);
    DTensor<T> A1(A, 2, 1, 1);
    A0.upload(aData);
    A1.upload(aData);
    std::vector<T> lowData = {3.162277660168380, 0, 0,
                              0.632455532033676, 4.427188724235731, 0,
                              0.948683298050514, -0.361403161162101, 5.382321781081287};  // from matlab
    DTensor<T> low(3, 3, 2);
    DTensor<T> low0(low, 2, 0, 0);
    DTensor<T> low1(low, 2, 1, 1);
    low0.upload(lowData, rowMajor);
    low1.upload(lowData, rowMajor);
    DTensor<T> L(low);
    CholeskyBatchFactoriser<T> chol(L, true);
    std::vector<T> bData = {-1., -3., 5.};
    DTensor<T> rhs(3, 1, 2);
    DTensor<T> rhs0(rhs, 2, 0, 0);
    DTensor<T> rhs1(rhs, 2, 1, 1);
    rhs0.upload(bData);
    rhs1.upload(bData);
    DTensor<T> sol(rhs);
    chol.solve(sol);
    std::vector<T> expected = {-0.126805213103205, -0.128566396618528, 0.175061641423036};
    std::vector<T> actual(6);
    sol.download(actual);
    for (size_t i = 0; i < 3; i++) EXPECT_NEAR(expected[i], actual[i], epsilon);  // 0
    for (size_t i = 0; i < 3; i++) EXPECT_NEAR(expected[i], actual[i + 3], epsilon);  // 1
    DTensor<T> error = A * sol;
    error -= rhs;
    EXPECT_TRUE(error.normF() < epsilon);
    EXPECT_EQ(0, chol.info()(0));
}

TEST_F(CholeskyTest, choleskyBatchSolve) {
    choleskyBatchSolve<float>(PRECISION_LOW);
    choleskyBatchSolve<double>(PRECISION_HIGH);
}


/* ================================================================================================
 *  QR TESTS
 * ================================================================================================ */
class QRTest : public testing::Test {
protected:
    QRTest() {}

    virtual ~QRTest() {}
};


/* ---------------------------------------
 * QR factorisation
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void qrFactorisation(T epsilon) {
    size_t nR = 4;
    size_t nC = 3;
    DTensor<T> temp(nR, nC);
    DTensor<T> A = DTensor<T>::createRandomTensor(nR, nC, 1, -100, 100);
    QRFactoriser<T> qr(temp);
    A.deviceCopyTo(temp);
    qr.factorise();
    EXPECT_EQ(0, qr.info()(0));
    DTensor<T> Q(nR, nC);
    DTensor<T> R(nC, nC, 1, true);
    DTensor<T> QR(nR, nC);
    qr.getQR(Q, R);
    EXPECT_EQ(0, qr.info()(0));
    QR.addAB(Q, R);
    QR -= A;
    T nrm = QR.normF();
    EXPECT_NEAR(nrm, 0., epsilon);
}

TEST_F(QRTest, qrFactorisation) {
    qrFactorisation<float>(PRECISION_LOW);
    qrFactorisation<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * QR factorisation
 * - tall and skinny matrix
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void qrFactorisationTall(T epsilon) {
    size_t nR = 20;
    size_t nC = 3;
    DTensor<T> temp(nR, nC);
    DTensor<T> A = DTensor<T>::createRandomTensor(nR, nC, 1, -100, 100);
    QRFactoriser<T> qr(temp);
    A.deviceCopyTo(temp);
    qr.factorise();
    EXPECT_EQ(0, qr.info()(0));
    DTensor<T> Q(nR, nC);
    DTensor<T> R(nC, nC, 1, true);
    DTensor<T> QR(nR, nC);
    qr.getQR(Q, R);
    EXPECT_EQ(0, qr.info()(0));
    QR.addAB(Q, R);
    QR -= A;
    T nrm = QR.normF();
    EXPECT_NEAR(nrm, 0., epsilon);
}

TEST_F(QRTest, qrFactorisationTall) {
    qrFactorisationTall<float>(PRECISION_LOW);
    qrFactorisationTall<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * QR factorisation: solve least squares
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void qrLeastSquares(T epsilon) {
    size_t nR = 4;
    size_t nC = 3;
    DTensor<T> temp(nR, nC);
    std::vector<T> vecA = {85.5638, -59.4001, -80.1992,
                           99.9464, 5.51393, 5.17935,
                           6.87488, -26.7536, 36.0914,
                           -44.3857, -32.1268, 54.8915};  // Random matrix
    std::vector<T> vecB = {-23.3585,
                           -48.5744,
                           43.4229,
                           -56.5081};  // Random vector
    DTensor<T> A(vecA, nR, nC, 1, rowMajor);
    DTensor<T> b(vecB, nR);
    DTensor<T> xFull(nR);
    DTensor<T> x(xFull, 0, 0, nC - 1);
    DTensor<T> Ax(nR);
    QRFactoriser<T> qr(temp);
    A.deviceCopyTo(temp);
    qr.factorise();
    EXPECT_EQ(0, qr.info()(0));
    b.deviceCopyTo(xFull);
    qr.leastSquares(xFull);
    EXPECT_EQ(0, qr.info()(0));
    Ax.addAB(A, x);
    Ax -= b;
    T nrm = Ax.normF();
    EXPECT_NEAR(nrm, 80.003169364198072, epsilon);  // From MatLab
}

TEST_F(QRTest, qrLeastSquares) {
    qrLeastSquares<float>(PRECISION_LOW);
    qrLeastSquares<double>(PRECISION_HIGH);
}


/* ================================================================================================
 *  NULLSPACE TESTS
 * ================================================================================================ */
class NullspaceTest : public testing::Test {
protected:
    NullspaceTest() {}

    virtual ~NullspaceTest() {}
};


/* ---------------------------------------
 * Basic nullspace test
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void computeNullspaceTensor(T epsilon) {
    std::vector<T> aData = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 0,
                            1, 2, 3, 4, 5, 6, 7, 8, 9, 7, 8, 9,
                            1, 2, 3, 4, 2, 4, 6, 8, 3, 6, 9, 12,
                            1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                            0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    DTensor<T> A(aData, 3, 4, 5);
    Nullspace<T> ns(A);
    DTensor<T> nA = ns.nullspace();
    size_t nMats = nA.numMats();
    EXPECT_EQ(nMats, 5);
    for (size_t i = 0; i < nMats; i++) {
        DTensor<T> nAi(nA, 2, i, i);
        DTensor<T> Ai(A, 2, i, i);
        DTensor<T> mustBeZero = Ai * nAi;
        EXPECT_LT(mustBeZero.normF(), epsilon);

        DTensor<T> nAiTr = nAi.tr();
        DTensor<T> mustBeEye = nAiTr * nAi;
        EXPECT_NEAR(1, mustBeEye(0, 0, 0), epsilon);
        for (size_t ir = 0; ir < mustBeEye.numRows(); ir++) {
            for (size_t ic = 0; ic < mustBeEye.numCols(); ic++) {
                if (ir != ic) {
                    EXPECT_NEAR(0, mustBeEye(ir, ic, 0), epsilon);
                }
            }
        }
    }
}

TEST_F(NullspaceTest, computeNullspaceTensor) {
    computeNullspaceTensor<float>(PRECISION_LOW);
    computeNullspaceTensor<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Nullspace is trivial
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void computeNullspaceTrivial(T epsilon) {
    std::vector<T> data{4, 5, 7,
                        4, 1, 8,
                        4, 5, 0,
                        1, 1, 1,
                        5, 6, 7,
                        9, 0, 3};
    DTensor<T> A(data, 3, 3, 2, rowMajor);
    Nullspace<T> nullA(A);
    DTensor<T> N = nullA.nullspace();
    EXPECT_EQ(N.normF(), 0);
}

TEST_F(NullspaceTest, computeNullspaceTrivial) {
    computeNullspaceTrivial<float>(PRECISION_LOW);
    computeNullspaceTrivial<double>(PRECISION_HIGH);
}

/* ---------------------------------------
 * Project onto nullspace
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void projectOnNullspaceTensor(T epsilon) {
    // offline
    size_t m = 3;
    size_t n = 7;
    std::vector<T> mat{1, -2, 3, 4, -1, -1, -1,
                       1, 2, -3, 4, -1, -1, -1,
                       -1, 3, 5, -7, -1, -1, -1};
    DTensor<T> A(m, n, 1);
    A.upload(mat, rowMajor);
    Nullspace<T> ns = Nullspace(A);
    DTensor<T> N = ns.nullspace();

    // online
    std::vector<T> vec{1, 2, 3, 4, 5, 6, 7};
    DTensor<T> x(vec, n);
    DTensor<T> proj(x);
    ns.project(proj);

    // Testing that proj is indeed in ker A
    DTensor<T> error(m, 1, 1, true);
    error.addAB(A, proj);
    EXPECT_TRUE(error.normF() < epsilon);

    // Orthogonality test (other - p) † (p - x)
    std::vector<T> h_other{1, -2, 5, 4, 0, 0, 0};
    DTensor<T> other(h_other, n);
    DTensor<T> y = N * other;
    DTensor<T> delta1 = y - proj;
    DTensor<T> delta2 = proj - x;
    EXPECT_LT(delta1.dotF(delta2), epsilon);
}

TEST_F(NullspaceTest, projectOnNullspaceTensor) {
    projectOnNullspaceTensor<float>(PRECISION_LOW);
    projectOnNullspaceTensor<double>(PRECISION_HIGH);
}


/* ================================================================================================
 *  GIVENSANNIHILATOR TESTS
 * ================================================================================================ */
class GivensAnnihilatorTest : public testing::Test {
protected:
    GivensAnnihilatorTest() {}

    virtual ~GivensAnnihilatorTest() {}
};


/* ---------------------------------------
 * GivensAnnihilator works
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void givensAnnihilateElement(T epsilon) {
    size_t m = 10;
    size_t n = 6;
    std::vector<T> v(m * n);
    v.reserve(m * n);
    std::iota(v.begin(), v.end(), 1);

    auto a = DTensor<T>(v, m, n, 1);
    auto ga = GivensAnnihilator<T>(a);
    size_t i = 0;
    for (size_t k = 1; k < m; k++) {
        for (size_t j = 0; j < n; j++) {
            ga.annihilate(i, k, j);
            EXPECT_NEAR(0.0, a(k, j), epsilon);
        }
    }
}

TEST_F(GivensAnnihilatorTest, givensAnnihilateElement) {
    givensAnnihilateElement<float>(PRECISION_LOW);
    givensAnnihilateElement<double>(PRECISION_HIGH);
}



/* ---------------------------------------
 * GivensAnnihilator: correctness
 * --------------------------------------- */

TEMPLATE_WITH_TYPE_T TEMPLATE_CONSTRAINT_REQUIRES_FPX
void givensAnnihilateCorrectness(T epsilon) {
    size_t m = 10, n = 6;
    std::vector<double> v(m * n);
    v.reserve(m * n);
    std::iota(v.begin(), v.end(), 1);
    DTensor<double> a = DTensor<double>(v, m, n);

    auto ga = GivensAnnihilator<double>(a);
    ga.annihilate(0, 1, 2);

    EXPECT_NEAR(0.0, a(1, 2), epsilon);
    EXPECT_NEAR(2.137186834969645, a(0, 0), epsilon);
    EXPECT_NEAR(44.552125559751822, a(0, 3), epsilon);
    EXPECT_NEAR(-0.328797974610715, a(1, 3), epsilon);

}

TEST_F(GivensAnnihilatorTest, givensAnnihilateCorrectness) {
    givensAnnihilateCorrectness<double>(1e-14);
    givensAnnihilateCorrectness<float>(1e-12);
}



